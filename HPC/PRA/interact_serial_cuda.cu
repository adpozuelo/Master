#include "hip/hip_runtime.h"
/*
 * Antonio Díaz Pozuelo - adpozuelo@uoc.edu
 * HPC_PRA - Energy-Potential N-Body Problem (Lennard Jones Interaction Potential)
 * CUDA vs CPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define NTHREAD 64
#define NDIM 3
#define SIDE 100

// CUDA kernel
// Energy per particle
__global__ void enerGPU(double eps4, double sig2, double *r, double *eng,
		int nmol) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < nmol) {
		int imol = i * NDIM;
		// Store particle i positions in register to avoid global memory access
		double xi = r[imol];
		double yi = r[imol + 1];
		double zi = r[imol + 2];
		double energ = 0;
		for (int j = 0; j < i; j++) {
			int jmol = j * NDIM;
			int jx = jmol;
			int jy = jmol + 1;
			int jz = jmol + 2;
			double dist2 = pow(xi - r[jx], 2) + pow(yi - r[jy], 2)
					+ pow(zi - r[jz], 2);
			double di6 = pow(sig2 / dist2, 3);
			double ulj = eps4 * di6 * (di6 - 1.0);
			energ = energ + ulj;
		}
		for (int j = i + 1; j < nmol; j++) {
			int jmol = j * NDIM;
			int jx = jmol;
			int jy = jmol + 1;
			int jz = jmol + 2;
			double dist2 = pow(xi - r[jx], 2) + pow(yi - r[jy], 2)
					+ pow(zi - r[jz], 2);
			double di6 = pow(sig2 / dist2, 3);
			double ulj = eps4 * di6 * (di6 - 1.0);
			energ = energ + ulj;
		}
		eng[i] = energ;
	}
}

// CPU kernel
// Energy of the system
void energia(double *energy, double eps4, double sig2, double *r, int nmol) {
	double energ = 0.0;
	int i, j;
	for (i = 0; i < nmol; i++) {
		int id = i * NDIM;
		double xi = r[id];
		double yi = r[id + 1];
		double zi = r[id + 2];
		for (j = i + 1; j < nmol; j++) {
			int jd = j * NDIM;
			int jx = jd;
			int jy = jd + 1;
			int jz = jd + 2;
			double dist2 = pow(xi - r[jx], 2) + pow(yi - r[jy], 2)
					+ pow(zi - r[jz], 2);
			double di6 = pow(sig2 / dist2, 3);
			double ulj = eps4 * di6 * (di6 - 1.0);
			energ = energ + ulj;
		}
	}
	*energy = energ;
}
int main(int argc, char *argv[]) {
	if (argc < 2) {
		printf("Usage: interact_serial NMOL\n");
		exit(-1);
	}
	int nmol = atoi(argv[1]);
	double *r, *energy;
	double *rdev, *edev;
	double eps = 120.0, sigma = 0.1, eps4, sig2, energ;
	clock_t start0, start_cpu, end;
	float cpu_time_used, totalcpu;
	// declare GPU timing events
	hipEvent_t start, stop, start_t, stop_t;
	int i, j;
	r = (double*) malloc(nmol * NDIM * sizeof(double));
	energy = (double*) malloc(nmol * sizeof(double));
	start0 = clock();
	eps4 = 4 * eps;
	sig2 = pow(sigma, 2);
	// Initialize particle positions (random)
	for (i = 0; i < nmol; i++) {
		for (j = 0; j < NDIM; j++) {
			r[i * NDIM + j] = SIDE * (double) rand() / (double) (RAND_MAX);
		}
	}
	// time CPU energy calculation
	start_cpu = clock();

	// Call CPU kernel for energy calculation
	energia(&energ, eps4, sig2, r, nmol);

	// time CPU results
	end = clock();
	cpu_time_used = 1000 * ((float) (end - start_cpu)) / CLOCKS_PER_SEC;
	totalcpu = 1000 * ((float) (end - start0)) / CLOCKS_PER_SEC;
	printf("%f ", cpu_time_used);
	printf("%f ", totalcpu);
	//printf(" Energy =%f\n", energ);

	// start GPU timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_t);
	hipEventCreate(&stop_t);
	hipEventRecord(start_t, 0);

	// Allocate memory on the GPU and transfer positions
	hipMalloc((void**) &rdev, NDIM * nmol * sizeof(double));
	hipMalloc((void**) &edev, nmol * sizeof(double));
	hipMemcpy(rdev, r, NDIM * nmol * sizeof(double), hipMemcpyHostToDevice);

	int nblock = nmol / NTHREAD;
	hipEventRecord(start, 0);

	// Call GPU kernel for energy calculation
	enerGPU<<<nblock, NTHREAD>>>(eps4, sig2, rdev, edev, nmol);

	// time GPU record
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Recall energy per particle from the GPU
	hipMemcpy(energy, edev, nmol * sizeof(double), hipMemcpyDeviceToHost);

	// Compute total energy
	double etot = 0;
	for (i = 0; i < nmol; i++) {
		etot = etot + energy[i];
	}
	etot = etot / 2;

	// total GPU time record
	hipEventRecord(stop_t, 0);
	hipEventSynchronize(stop_t);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	float elapsedTimet;
	hipEventElapsedTime(&elapsedTimet, start_t, stop_t);

	printf("%f ", elapsedTime);
	printf("%f\n", elapsedTimet);
	//printf(" GPU Energy =%f\n", etot);
}
